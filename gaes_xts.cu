#include "hip/hip_runtime.h"
/**
 * Reference:
 * 1. https://nvlpubs.nist.gov/nistpubs/fips/nist.fips.197.pdf
 * 2. https://en.wikipedia.org/wiki/Advanced_Encryption_Standard
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "gaes.cu"

#define XTS_SECTOR_SIZE (512)

#define gf128mul_dat(q)                                                    \
    {                                                                      \
        q(0x00), q(0x01), q(0x02), q(0x03), q(0x04), q(0x05), q(0x06),     \
            q(0x07), q(0x08), q(0x09), q(0x0a), q(0x0b), q(0x0c), q(0x0d), \
            q(0x0e), q(0x0f), q(0x10), q(0x11), q(0x12), q(0x13), q(0x14), \
            q(0x15), q(0x16), q(0x17), q(0x18), q(0x19), q(0x1a), q(0x1b), \
            q(0x1c), q(0x1d), q(0x1e), q(0x1f), q(0x20), q(0x21), q(0x22), \
            q(0x23), q(0x24), q(0x25), q(0x26), q(0x27), q(0x28), q(0x29), \
            q(0x2a), q(0x2b), q(0x2c), q(0x2d), q(0x2e), q(0x2f), q(0x30), \
            q(0x31), q(0x32), q(0x33), q(0x34), q(0x35), q(0x36), q(0x37), \
            q(0x38), q(0x39), q(0x3a), q(0x3b), q(0x3c), q(0x3d), q(0x3e), \
            q(0x3f), q(0x40), q(0x41), q(0x42), q(0x43), q(0x44), q(0x45), \
            q(0x46), q(0x47), q(0x48), q(0x49), q(0x4a), q(0x4b), q(0x4c), \
            q(0x4d), q(0x4e), q(0x4f), q(0x50), q(0x51), q(0x52), q(0x53), \
            q(0x54), q(0x55), q(0x56), q(0x57), q(0x58), q(0x59), q(0x5a), \
            q(0x5b), q(0x5c), q(0x5d), q(0x5e), q(0x5f), q(0x60), q(0x61), \
            q(0x62), q(0x63), q(0x64), q(0x65), q(0x66), q(0x67), q(0x68), \
            q(0x69), q(0x6a), q(0x6b), q(0x6c), q(0x6d), q(0x6e), q(0x6f), \
            q(0x70), q(0x71), q(0x72), q(0x73), q(0x74), q(0x75), q(0x76), \
            q(0x77), q(0x78), q(0x79), q(0x7a), q(0x7b), q(0x7c), q(0x7d), \
            q(0x7e), q(0x7f), q(0x80), q(0x81), q(0x82), q(0x83), q(0x84), \
            q(0x85), q(0x86), q(0x87), q(0x88), q(0x89), q(0x8a), q(0x8b), \
            q(0x8c), q(0x8d), q(0x8e), q(0x8f), q(0x90), q(0x91), q(0x92), \
            q(0x93), q(0x94), q(0x95), q(0x96), q(0x97), q(0x98), q(0x99), \
            q(0x9a), q(0x9b), q(0x9c), q(0x9d), q(0x9e), q(0x9f), q(0xa0), \
            q(0xa1), q(0xa2), q(0xa3), q(0xa4), q(0xa5), q(0xa6), q(0xa7), \
            q(0xa8), q(0xa9), q(0xaa), q(0xab), q(0xac), q(0xad), q(0xae), \
            q(0xaf), q(0xb0), q(0xb1), q(0xb2), q(0xb3), q(0xb4), q(0xb5), \
            q(0xb6), q(0xb7), q(0xb8), q(0xb9), q(0xba), q(0xbb), q(0xbc), \
            q(0xbd), q(0xbe), q(0xbf), q(0xc0), q(0xc1), q(0xc2), q(0xc3), \
            q(0xc4), q(0xc5), q(0xc6), q(0xc7), q(0xc8), q(0xc9), q(0xca), \
            q(0xcb), q(0xcc), q(0xcd), q(0xce), q(0xcf), q(0xd0), q(0xd1), \
            q(0xd2), q(0xd3), q(0xd4), q(0xd5), q(0xd6), q(0xd7), q(0xd8), \
            q(0xd9), q(0xda), q(0xdb), q(0xdc), q(0xdd), q(0xde), q(0xdf), \
            q(0xe0), q(0xe1), q(0xe2), q(0xe3), q(0xe4), q(0xe5), q(0xe6), \
            q(0xe7), q(0xe8), q(0xe9), q(0xea), q(0xeb), q(0xec), q(0xed), \
            q(0xee), q(0xef), q(0xf0), q(0xf1), q(0xf2), q(0xf3), q(0xf4), \
            q(0xf5), q(0xf6), q(0xf7), q(0xf8), q(0xf9), q(0xfa), q(0xfb), \
            q(0xfc), q(0xfd), q(0xfe), q(0xff)                             \
    }

#define xx(p, q) 0x##p##q

#define xda_bbe(i)                                               \
    ((i & 0x80 ? xx(43, 80) : 0) ^ (i & 0x40 ? xx(21, c0) : 0) ^ \
     (i & 0x20 ? xx(10, e0) : 0) ^ (i & 0x10 ? xx(08, 70) : 0) ^ \
     (i & 0x08 ? xx(04, 38) : 0) ^ (i & 0x04 ? xx(02, 1c) : 0) ^ \
     (i & 0x02 ? xx(01, 0e) : 0) ^ (i & 0x01 ? xx(00, 87) : 0))

__constant__ uint16_t gf128mul_table_bbe[256] = gf128mul_dat(xda_bbe);

#define gf128mul_x_ble(r, x)                                  \
    (*r = gf128mul_table_bbe[(*(x + 1)) >> 63] ^ ((*x) << 1), \
     *(r + 1) = ((*(x + 1)) << 1) ^ ((*x) >> 63))

#define be128_xor(r, p, q) ((r)[0] = (p)[0] ^ (q)[0], (r)[1] = (p)[1] ^ (q)[1])

__device__ void xts_encrypt(uint8_t* key,
                            uint32_t key_len,
                            uint8_t* data,
                            const uint64_t tweak) {
    unsigned int i;
    uint32_t nrounds = key_len / 2 / 4 + 6;
    uint64_t tweak_buf[AES_BLOCK_SIZE / sizeof(uint64_t)] = {tweak + blockIdx.x,
                                                             0};

    data = data + AES_BLOCK_SIZE * (blockIdx.x * blockDim.x + threadIdx.x);

    /* calculate first value of T */
    aes_encrypt(nrounds, key + key_len / 2, (uint8_t*)tweak_buf);

    for (i = 1; i <= threadIdx.x; i++) {
        gf128mul_x_ble(tweak_buf, tweak_buf);
    }

    /* PP <- T xor P */
    be128_xor((uint64_t*)data, tweak_buf, (uint64_t*)data);
    /* CC <- E(Key2,PP) */
    aes_encrypt(nrounds, key, data);
    /* C <- C xor CC */
    be128_xor((uint64_t*)data, (uint64_t*)data, tweak_buf);
}

__device__ void xts_decrypt(uint8_t* key,
                            uint32_t key_len,
                            uint8_t* data,
                            const uint64_t tweak) {
    unsigned int i;
    uint32_t nrounds = key_len / 2 / 4 + 6;
    uint64_t tweak_buf[AES_BLOCK_SIZE / sizeof(uint64_t)] = {tweak + blockIdx.x,
                                                             0};

    data = data + AES_BLOCK_SIZE * (blockIdx.x * blockDim.x + threadIdx.x);

    /* calculate first value of T */
    aes_encrypt(nrounds, key + key_len / 2, (uint8_t*)tweak_buf);

    for (i = 1; i <= threadIdx.x; i++) {
        gf128mul_x_ble(tweak_buf, tweak_buf);
    }

    /* PP <- T xor P */
    be128_xor((uint64_t*)data, tweak_buf, (uint64_t*)data);
    /* CC <- E(Key2,PP) */
    aes_decrypt(nrounds, key, data);
    /* C <- C xor CC */
    be128_xor((uint64_t*)data, (uint64_t*)data, tweak_buf);
}

#ifndef GXTS_HMAC

__global__ void global_xts_encrypt(uint8_t* key,
                                   uint32_t key_len,
                                   uint8_t* data,
                                   const uint64_t tweak) {
    xts_encrypt(key, key_len, data, tweak);
}

__global__ void global_xts_decrypt(uint8_t* key,
                                   uint32_t key_len,
                                   uint8_t* data,
                                   const uint64_t tweak) {
    xts_decrypt(key, key_len, data, tweak);
}

#define TEXT_SIZE (4 * 1024 * 1024)
#define KEY_LENGTH (64)

int main(int argc, char const* argv[]) {
    uint8_t* dev_text;
    uint8_t* text = (uint8_t*)malloc(TEXT_SIZE);
    uint8_t* dev_key;
    uint8_t key[KEY_LENGTH] = {
        0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09, 0x0a,
        0x0b, 0x0c, 0x0d, 0x0e, 0x0f, 0x10, 0x11, 0x12, 0x13, 0x14, 0x15,
        0x16, 0x17, 0x18, 0x19, 0x1a, 0x1b, 0x1c, 0x1d, 0x1e, 0x1f, 0x20,
        0x21, 0x22, 0x23, 0x24, 0x25, 0x26, 0x27, 0x28, 0x29, 0x2a, 0x2b,
        0x2c, 0x2d, 0x2e, 0x2f, 0x30, 0x31, 0x32, 0x33, 0x34, 0x35, 0x36,
        0x37, 0x38, 0x39, 0x3a, 0x3b, 0x3c, 0x3d, 0x3e, 0x3f};

    uint32_t key_bit[3] = {128, 192, 256};
    memset(text, 1, TEXT_SIZE);

    hipMalloc((void**)&dev_text, TEXT_SIZE);
    hipMalloc((void**)&dev_key, sizeof(key));
    hipMemcpy(dev_text, text, TEXT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_key, key, sizeof(key), hipMemcpyHostToDevice);

    // one aes_block per thread
    dim3 dimBlock(XTS_SECTOR_SIZE / AES_BLOCK_SIZE, 1);
    // one xts_sector per block
    dim3 dimGrid(TEXT_SIZE / XTS_SECTOR_SIZE, 1);

    uint32_t i;
    for (i = 0; i < sizeof(key_bit) / sizeof(key_bit[0]); ++i) {
        printf("key_bit %d: \n", key_bit[i]);
        printf(
            "\tinput  :   0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x "
            "0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x\n",
            text[0], text[1], text[2], text[3], text[4], text[5], text[6],
            text[7], text[8], text[9], text[10], text[11], text[12], text[13],
            text[14], text[15]);

        global_xts_encrypt<<<dimGrid, dimBlock>>>(dev_key, key_bit[i] * 2 / 8,
                                                  dev_text, 0);
        hipMemcpy(text, dev_text, TEXT_SIZE, hipMemcpyDeviceToHost);
        printf(
            "\tencrypt  :   0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x "
            "0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x\n",
            text[0], text[1], text[2], text[3], text[4], text[5], text[6],
            text[7], text[8], text[9], text[10], text[11], text[12], text[13],
            text[14], text[15]);

        global_xts_decrypt<<<dimGrid, dimBlock>>>(dev_key, key_bit[i] * 2 / 8,
                                                  dev_text, 0);
        hipMemcpy(text, dev_text, TEXT_SIZE, hipMemcpyDeviceToHost);
        printf(
            "\tdecrypt  :   0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x "
            "0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x 0x%.2x\n",
            text[0], text[1], text[2], text[3], text[4], text[5], text[6],
            text[7], text[8], text[9], text[10], text[11], text[12], text[13],
            text[14], text[15]);
    }
    return 0;
}

#endif
